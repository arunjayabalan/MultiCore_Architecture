
#include <hip/hip_runtime.h>
#include <assert.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>

#define HEIGHT 256
#define WIDTH 256
#define IMAGE_SIZE 256*256
#define IMAGE 256
#define THREAD_SIZE1 8
#define THREAD_SIZE2 8 




__device__ int iterate_pixel(float x, float y, float c_re, float c_im)
{
	int c=0;
	float z_re=x;
	float z_im=y;
	while (c < 255) {
		float re2 = z_re*z_re;
		float im2 = z_im*z_im;
		if ((re2+im2) > 4) 
			break; 
		z_im=2*z_re*z_im + c_im;
		z_re=re2-im2 + c_re;
		c++;
	}
	return c;
}

//calc_fractal<<< N, N >>>( 28, 0.008, fractal);
__global__ void calc_fractal(float c_re, float c_im, unsigned char *fractal)
{
	
//blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y +  threadIdx.y ) *( blockDim.x * gridDim.x)
//(blockIdx.x + blockIdx.y * gridDim.x) * (blockDim.x * blockDim.y) + threadIdx.y * blockDim.x + threadIdx.x 



	int width = 256, height = 256;
	int x = ((blockIdx.x * blockDim.x) + threadIdx.x + (blockIdx.y * blockDim.y +  threadIdx.y ) *( blockDim.x * gridDim.x) )%HEIGHT;  
	int y = (blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y +  threadIdx.y ) *( blockDim.x * gridDim.x))/WIDTH;
		  
	float f_x = (float)(x*0.8)/(float)(width)-0.8;
	float f_y = (float)(y*0.8)/(float)(height)-0.8;
	fractal[blockIdx.x * blockDim.x + threadIdx.x + (blockIdx.y * blockDim.y +  threadIdx.y ) *( blockDim.x * gridDim.x) ] = iterate_pixel(f_x,f_y,c_re,c_im);

}

// Write a width by height 8-bit color image into File "filename"
void write_ppm(unsigned char* data,unsigned int width,unsigned int height,char* filename)
{
	if (data == NULL) {
		printf("Provide a valid data pointer!\n");
		return;
	}
	if (filename == NULL) {
		printf("Provide a valid filename!\n");
		return;
	}
	if ( (width>4096) || (height>4096)) {
		printf("Only pictures upto 4096x4096 are supported!\n");
		return;
	}
	FILE *f=fopen(filename,"wb");
	if (f == NULL) 
	{
		printf("Opening File %s failed!\n",filename);
		return;
	}
	if (fprintf(f,"P6 %i %i 255\n",width,height) <= 0) {
		printf("Writing to file failed!\n");
		return;
	};
	int i;
	for (i=0;i<height;i++) {
		unsigned char buffer[4096*3];
		int j;
		for (j=0;j<width;j++) {
			int v=data[i*width+j];
			int s;
			s= v << 0;
			s=s > 255? 255 : s;
			buffer[j*3+0]=s;
			s= v << 1;
			s=s > 255? 255 : s;
			buffer[j*3+1]=s;
			s= v << 2;
			s=s > 255? 255 : s;
			buffer[j*3+2]=s;
		}
		if (fwrite(buffer,width*3,1,f) != 1) {
			printf("Writing of line %i to file failed!\n",i);
			return;
		}
	}
	fclose(f);
}

int main(int argc, char** args)
{
	int N = 256*256;
	unsigned char julia[N];
	unsigned char* fractal;


	for(int i = 0; i < N; i++)
	{

		julia[i] = i;
	}
	
	// memory allocation
	hipMalloc ( (void**)&fractal, (N)*sizeof(char) );

	//copying from host to Device
	hipMemcpy(fractal, &julia, (N)*sizeof(char), hipMemcpyHostToDevice);
	
	dim3 numBlocks(IMAGE/THREAD_SIZE1, IMAGE/THREAD_SIZE2); 

	dim3 numThreads(THREAD_SIZE1, THREAD_SIZE2);	
 
	//Kernel invocation
	calc_fractal<<< numBlocks, numThreads>>>( 0.28, 0.008, fractal);
	

	//copying from Device to Host
	hipMemcpy(&julia, fractal, (N)*sizeof(char), hipMemcpyDeviceToHost) ;

	write_ppm(julia, 256, 256, "julia2D.ppm");	
	
	hipFree(&fractal);

	
	return 0;
}
